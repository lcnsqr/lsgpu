#include <stdio.h>
#include "gpu.h"

// Cuda cores per multiprocessor from Compute Capability
inline int _ConvertSMVer2Cores(int major, int minor) {
  // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
  typedef struct {
    int SM; // 0xMm (hexadecimal notation), M = SM Major version, and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
    { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
    { 0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
    { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
    { 0x37, 192}, // Kepler Generation (SM 3.7) GK21x class
    { 0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
    { 0x52, 128}, // Maxwell Generation (SM 5.2) GM20x class
    { 0x53, 128}, // Maxwell Generation (SM 5.3) GM20x class
    { 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
    { 0x61, 128}, // Pascal Generation (SM 6.1) GP10x class
    { 0x62, 128}, // Pascal Generation (SM 6.2) GP10x class
    { 0x70, 64 }, // Volta Generation (SM 7.0) GV100 class
		{ 0x72, 64 }, // Xavier Generation (SM 7.2) GV10B class
		{ 0x75, 64 }, // Turing Generation (SM 7.5) TU102 class
		{ 0x80, 64 }, // Ampere Generation (SM 8.0) GA10x class
    {   -1, -1 }
  };
  int index = 0;
  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)){
      return nGpuArchCoresPerSM[index].Cores;
    }
    index++;
  }
  // If we don't find the values, we default use the previous one to run properly
  //fprintf(stderr, "MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
  return nGpuArchCoresPerSM[index-1].Cores;
}

// How many local GPUs
int gpu_count(){
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess){
    //fprintf(stderr, "Error: hipGetDeviceCount returns %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
    deviceCount = -1;
  }
  return deviceCount;
}

// Load GPU properties
int gpu_properties(hipDeviceProp_t* deviceProp, int deviceCount){
  hipError_t error_id;
  for (int c = 0; c < deviceCount; c++){
    error_id = hipGetDeviceProperties(&deviceProp[c], c);
    if (error_id != hipSuccess) return -1;
  }
  return deviceCount;
}

// Descriptive GPU details
int gpu_description(char* buffer){

  int deviceCount = gpu_count();
  if ( deviceCount < 0 ) return -1;

  hipDeviceProp_t *deviceProp = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t) * deviceCount);
  gpu_properties(deviceProp, deviceCount);
  
	int driverVersion = 0;
	int runtimeVersion = 0;

  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);

  // GPU chipset family
  char family[32];

  // Description entry
  char buf_entry[4096];

  for (int c = 0; c < deviceCount && c < MAX_GPU; c++){

    // Get GPU family
    switch ( deviceProp[c].major ){
      case 3: sprintf(family, "Kepler"); break;
      case 5: sprintf(family, "Maxwell"); break;
      case 6: sprintf(family, "Pascal"); break;
      case 7:
        if ( deviceProp[c].minor == 0 )
          sprintf(family, "Volta");
        if ( deviceProp[c].minor == 2 )
          sprintf(family, "Xavier");
        if ( deviceProp[c].minor == 5 )
          sprintf(family, "Turing");
      break;
      case 8: sprintf(family, "Ampere"); break;
    }

    snprintf(buf_entry, 4096, "\
  Device:          %d\n\
  Name:            %s\n\
  Family:          %s\n\
  Capability:      %d.%d\n\
  Cores / MP:      %d\n\
  Global Memory:   %.0f MB\n\
  Driver:          %d.%d\n\
  Runtime:         %d.%d\n", 
    c,
    deviceProp[c].name,
    family,
    deviceProp[c].major, deviceProp[c].minor,
    _ConvertSMVer2Cores(deviceProp[c].major, deviceProp[c].minor),
    (float)deviceProp[c].totalGlobalMem/1048576.0f,
    driverVersion/1000, (driverVersion%100)/10,
    runtimeVersion/1000, (runtimeVersion%100)/10);

    // Add to description
    strncat(buffer, buf_entry, 4096);
  }

  free(deviceProp);

  return deviceCount;
}
